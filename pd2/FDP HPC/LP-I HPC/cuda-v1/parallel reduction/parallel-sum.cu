#include "hip/hip_runtime.h"

#include<bits/stdc++.h>
#include <iostream>
#include <numeric>

using namespace std;

#define BLOCK_SIZE 4

__global__ void sum(int* input)  		// kernel function definition
{
	const int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;    // blockDim = 4 i.e. number of threads per block = 4
	

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{     
			const int fst = tid * step_size * 2;    //get the index in array
			const int snd = fst + step_size;		//get the index in array
			input[fst] += input[snd];				//calculate sum
					
		}

		step_size <<= 1; 				// increment step_size by 1
		number_of_threads >>= 1;		//decrement number of threads by 2
	}
	
}

int main()
{
	int count=0;
	cout << "Enter the number of elements:\n" << endl;
	cin>>count;

	const int size = count * sizeof(int);
	
	int h[count];
	cout << "Enter the elements:\n" << endl;
	for(int i=0;i<count;i++)
	{
		cout << "\n";
		cin >> h[i];
	}
	

	int* d;							//GPU parameter
       
	
	hipMalloc(&d, size);			//assign memory to parameters on GPU
	
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);		//copy the array from CPU to GPU
	sum <<<1, count / 2 >>>(d);							// call kernel function <<<number of blocks, number of threads= number of elements/2
	
	int result;
	
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);		// copy the result back from GPU to CPU

	cout << "Sum is " << result << endl;

	getchar();

	hipFree(d);					// Free the allocated memory


	return 0;
}
